
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>

void matrixMulCPU(float *C, const float *A, const float *B, unsigned int hA, unsigned int wA, unsigned int wB)
{
	for (int i = 0; i < hA; ++i)
	{
		for (int j = 0; j < wB; ++j)
		{
			float sum = 0;
			for (int k = 0; k < wA; ++k)
			{
				sum += A[i * wA + k] * B[k * wB + j];
			}
			C[i * wB + j] = sum;
		}
	}
}

int main(int argc, char *argv[])
{
	// Initialize constants.
	int block_size = 32;
	int uiWA = 2 * block_size * 5;
	int uiHA = 4 * block_size * 5;
	int uiWB = 2 * block_size * 5;
	int uiHB = 4 * block_size * 5;
	int uiWC = 2 * block_size * 5;
	int uiHC = 4 * block_size * 5;
	unsigned int size_A = uiWA * uiHA;
	unsigned int size_B = uiWB * uiHB;
	unsigned int size_C = uiWC * uiHC;
	unsigned int mem_size_A = sizeof(float) * size_A;
	unsigned int mem_size_B = sizeof(float) * size_B;
	unsigned int mem_size_C = sizeof(float) * size_C;

	// Allocates matrices a, b and c in host memory.
	float *h_A = (float *)malloc(mem_size_A);
	float *h_B = (float *)malloc(mem_size_B);
	float *h_C = (float *)malloc(mem_size_C);

	// Initialize matrices a and b.
	srand(2006);
	for (int i = 0; i < size_A; ++i)
	{
		h_A[i] = rand() / (float)RAND_MAX;
	}
	for (int i = 0; i < size_B; ++i)
	{
		h_B[i] = rand() / (float)RAND_MAX;
	}

	// Allocate matrices a, b and c in device memory.
	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, mem_size_A);
	hipMalloc((void **)&d_B, mem_size_B);
	hipMalloc((void **)&d_C, mem_size_C);

	// Copy matrices a and b from host memory to device memory.
	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

	// Determine the number of threads per block and the number of blocks per grid.
	dim3 numThreadsPerBlock(block_size, block_size);
	dim3 numBlocksPerGrid(uiWC / numThreadsPerBlock.x, uiHC / numThreadsPerBlock.y);

	// Initialize a cublas handle.
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// CUBLAS is column primary.
	const float alpha = 1.0f;
	const float beta  = 0.0f;
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, uiWB, uiHA, uiWA, &alpha, d_B, uiWB, d_A, uiWA, &beta, d_C, uiWA);

	// Measure the performance of cublasSgemm over a number of iterations.
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	int numIterations = 30;
	for (int i = 0; i < numIterations; ++i)
	{
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, uiWB, uiHA, uiWA, &alpha, d_B, uiWB, d_A, uiWA, &beta, d_C, uiWA);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsed;
	hipEventElapsedTime(&elapsed, start, stop);

	// Compute and print the GLOPS/s performance metric.
	printf("%.2f GFLOP/s\n", (2.0f * uiWA * uiHA * uiWB * numIterations * 1e-9f) / (elapsed / 1000.0f));

	// Copy matrix c from device memory to host memory.
	hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

	// Compute reference solution.
	float *ref = (float *)malloc(mem_size_C);
	matrixMulCPU(ref, h_A, h_B, uiHA, uiWA, uiWB);

	// Validate the result.
	for (int i = 0; i < size_C; ++i)
	{
		float actual = h_C[i];
		float expected = ref[i];
		if (fabs(actual - expected) / fabs(actual) / uiWA > 1e-7)
		{
			printf("h_C[%d] = %f, expected = %f\n", i, actual, expected);
			break;
		}
	}

	// Cleanup.
	free(ref);
	hipblasDestroy(handle);
	hipFree(d_C);
	hipFree(d_B);
	hipFree(d_A);
	hipDeviceReset();
	free(h_C);
	free(h_B);
	free(h_A);
}
